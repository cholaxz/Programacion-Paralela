
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

#define SIZE 150 * 1000

#define THREADS 256 //best value = 256

#define SORT 1
#define TestReduction 0
#define PRINT 1
#define printErrors 1
#define CHECK 1
#define DATATYPE struct number
#define VALUETYPE int
#define RECORDTIME 1
#define MIN INT_MIN

#define OPTION 2
/*
1: i+1 
2: SIZE-i
3: rand() % 100
*/

//#define CUDA_ERROR_CHECK

/* Function declarations */
void getGridComposition(int, unsigned int *, unsigned int *);
void printResults(VALUETYPE *);

/* Struct for not losing the global index */
struct number
{
    VALUETYPE value;
    unsigned int index;
};

/* Error Checking */

#define CudaSafeCall(err) __cudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)

inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
    if (cudaSuccess != err)
    {
        fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, cudaGetErrorString(err));
        exit(-1);
    }
#endif
    return;
}

inline void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
    cudaError err = cudaGetLastError();
    if (cudaSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line, cudaGetErrorString(err));
        exit(-1);
    }

    /* Can affect performance. Comment if needed. */
    err = cudaDeviceSynchronize();
    if (cudaSuccess != err)
    {
        fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file, line, cudaGetErrorString(err));
        exit(-1);
    }
#endif
    return;
}

__global__ void loadKernel(int size, DATATYPE * g_list, DATATYPE * g_loadto){
    unsigned int tid = threadIdx.x;
    unsigned int gid = (blockIdx.x * blockDim.x) + tid;

    if(gid < size){
        g_loadto[gid] = g_list[gid];
    }
}
/* Kernel reduction at block level */
/* One thread per data */
__global__ void reduceKernel(int size, DATATYPE *g_input, DATATYPE *g_output)
{

    unsigned int tid = threadIdx.x;
    unsigned int gid = (blockIdx.x * blockDim.x) + tid;

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if (gid + s < size)
                g_input[gid] = g_input[gid].value > g_input[gid + s].value ? g_input[gid] : g_input[gid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
        g_output[blockIdx.x] = g_input[gid];
}

/* This function swaps the MAX element [which should be in position 0] with the last element of the list */
/* WARNING: This function must be called by only one block */
__global__ void swapKernel(int size, DATATYPE *g_list, DATATYPE *g_max)
{
    DATATYPE max;
    DATATYPE last_element;
    unsigned int index;

    max = g_max[0];
    last_element = g_list[size - 1];

    index = max.index;
    max.index = size - 1;
    last_element.index = index;

    g_list[index] = last_element; /* Donde estaba el valor maximo, pongo el ultimo elemento de la lista */
    g_list[size - 1] = max;     /* Al maximo lo pongo al final de la lista */
    
}

/* Kernel call that wraps data into a struct with index */
__global__ void wrapKernel(int size, DATATYPE *g_wrapped_list, VALUETYPE *g_list)
{
    unsigned int gid = threadIdx.x + blockDim.x * blockIdx.x;
    DATATYPE myData;

    if (gid < size)
    {
        myData.value = g_list[gid];
        myData.index = gid;
        g_wrapped_list[gid] = myData;
    }
}

/* Kernel call that unwraps data into an array of VALUETYPE */
__global__ void unwrapKernel(int size, DATATYPE *g_wrapped_list, VALUETYPE *g_list)
{
    unsigned int gid = threadIdx.x + blockDim.x * blockIdx.x;

    if (gid < size)
    {
        g_list[gid] = g_wrapped_list[gid].value;
    }
}

/* Wraps Reduction Kernel Call */
DATATYPE * reduceMax(int size, DATATYPE *g_list, DATATYPE *g_wa, DATATYPE *g_wb)
{

    unsigned int threads, blocks;
    DATATYPE *input, *output, *ptr;

    int N, temp;
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);

    getGridComposition(size, &blocks, &threads);
    dimGrid.x = blocks;
    dimBlock.x = threads;

    N = size;

    input = g_wa;
    output = g_wb;

    loadKernel<<<dimGrid, dimBlock>>>(N, g_list, input);

    while (dimGrid.x > 0)
    {
        //printf("Bloques: %d, N: %d\n", dimGrid.x, N);
        reduceKernel<<<dimGrid, dimBlock>>>(N, input, output);
        CudaCheckError();

        temp = (N / dimBlock.x);
        if (N % (dimBlock.x) != 0)
            temp++;
        N = temp;
        
        ptr = input;
        input = output;
        output = ptr;

        if (dimGrid.x == 1)
            dimGrid.x = 0;
        else
            dimGrid.x = dimGrid.x > dimBlock.x ? dimGrid.x / dimBlock.x : 1;
    }

    return input; /* At this point input is the last output */
}

/* Calls the iterative reduction wrapper and sorts the max results */
void sortBySelectionIterative(int size, DATATYPE *g_wlist, DATATYPE *g_wa, DATATYPE *g_wb)
{
    DATATYPE * result;
    for (int i = size; i > 1; i--)
    {
        result = reduceMax(i, g_wlist, g_wa, g_wb);
        swapKernel<<<1, 1>>>(i, g_wlist, result);

        /* Test */
        /*
        unwrapKernel<<<1, 256>>>(SIZE, g_list, g_test_list);
        CudaCheckError();
        CudaSafeCall(cudaMemcpy(test_list, g_list, SIZE * sizeof(VALUETYPE), cudaMemcpyDeviceToHost));
        printResults(test_list);
        */

    }

    return;
}

/* Get the number of blocks and threads per block */
void getGridComposition(int size, unsigned int *blocks, unsigned int *threads)
{

    *threads = THREADS;
    *blocks = 1;

    while (((*blocks) * (*threads)) < size)
    {
        *blocks <<= 1;
    }

    return;
}

void printResults(VALUETYPE *sorted_list)
{
    if (printErrors)
    {
        for (int i = 0; i < SIZE; i++)
        {
            if (sorted_list[i] != (i + 1))
                printf("%d: %d \n", i + 1, sorted_list[i]);
        }
        printf("\n");
    }
    else
    {
        for (int i = 0; i < SIZE; i++)
        {
            printf("%d\n", sorted_list[i]);
        }
        printf("\n");
    }

    return;
}

int checkResults(VALUETYPE *sorted_list)
{
    unsigned int check = 1;
    unsigned int i;
    for (i = 0; i < SIZE; i++)
    {
        if (sorted_list[i] != (i + 1))
            check = 0;
    }

    if (check)
        printf("Resultados correctos!\n");
    else
        printf("Resultados incorrectos!\n");

    return check;
}

int main(void)
{
    printf("SIZE: %d\n", SIZE);
    DATATYPE *g_wlist, *g_wa, *g_wb;
    VALUETYPE *list, *g_list;

    srand(time(NULL));
    /* Allocate Host memory */
    list = (VALUETYPE *)malloc(SIZE * sizeof(VALUETYPE));
    if (list == NULL)
    {
        printf("Error alocando memoria.\n");
        exit(-1);
    }

    /* Allocate device memory */
    CudaSafeCall(hipMalloc((void **)&g_list, SIZE * sizeof(VALUETYPE)));
    CudaSafeCall(hipMalloc((void **)&g_wlist, SIZE * sizeof(DATATYPE)));
    CudaSafeCall( hipMalloc((void**)&g_wa, SIZE  * sizeof(DATATYPE) ) );

    CudaSafeCall( hipMalloc((void**)&g_wb, (SIZE / THREADS + 1)  * sizeof(DATATYPE) ) );

    /* Initialize data */
    for (int i = 0; i < SIZE; i++)
    {
        switch (OPTION)
        {
        case 1:
            list[i] = i + 1;
            break;
        case 2:
            list[i] = SIZE - i;
            break;
        case 3:
            list[i] = rand() % 100;
            break;
        }        
    }

    /* Wrap Data into a struct with index for sorting */
    unsigned int threads, blocks;
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(1, 1, 1);

    CudaSafeCall( hipMemcpy(g_list, list, SIZE * sizeof(VALUETYPE), hipMemcpyHostToDevice) );

    getGridComposition(SIZE, &blocks, &threads);
    dimGrid.x = blocks;
    dimBlock.x = threads;

    wrapKernel<<<dimGrid, dimBlock>>>(SIZE, g_wlist, g_list );
    CudaCheckError();
    /* End of wrapping data */


    if (SORT){
        /* Record time */
        hipEvent_t start, stop;
        if (RECORDTIME)
        {
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
        }
        sortBySelectionIterative(SIZE, g_wlist, g_wa, g_wb);
        unwrapKernel<<<dimGrid, dimBlock>>>(SIZE, g_wlist, g_list);
        if (RECORDTIME)
        {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Pasaron %f milisegundos\n", milliseconds);
        }

        CudaCheckError();
        CudaSafeCall(hipMemcpy(list, g_list, SIZE * sizeof(VALUETYPE), hipMemcpyDeviceToHost));
    } 

    if (TestReduction){
        /* Record time */
        hipEvent_t start, stop;
        if (RECORDTIME)
        {
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
        }

        reduceMax(SIZE, g_wlist, g_wa, g_wb);

        if (RECORDTIME)
        {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("Pasaron %f milisegundos\n", milliseconds);
        }

        unwrapKernel<<<1, 1>>>(1, g_wb, g_list);
        CudaCheckError();
        CudaSafeCall(hipMemcpy(list, g_list, 1 * sizeof(VALUETYPE), hipMemcpyDeviceToHost));
    } 
    

    if(TestReduction){
        printf("El maximo es %d\n", list[0]);
    }

    if (PRINT && SORT)
    {
        printResults(list);
    }
    if(CHECK && SORT)
    {
        checkResults(list);
    }

    CudaSafeCall ( hipFree(g_list) );
    CudaSafeCall ( hipFree(g_wa) );
    CudaSafeCall ( hipFree(g_wb) );
    CudaSafeCall ( hipFree(g_wlist) );

    free(list);
}